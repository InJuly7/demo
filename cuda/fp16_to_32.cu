#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

__global__ void printFP16Values(const __half* input) {
    int idx = threadIdx.x;
    printf("Thread %d: FP16 value = %f\n", idx, __half2float(input[idx]));
}

int main() {
    const int size = 4;
    float h_data[size] = {1.5f, 2.75f, -3.25f, 4.0f};
    __half* h_fp16 = new __half[size];

    for(int i = 0; i < size; i++) {
        h_fp16[i] = __float2half(h_data[i]);
    }

    __half* d_fp16;
    hipMalloc(&d_fp16, size * sizeof(__half));
    hipMemcpy(d_fp16, h_fp16, size * sizeof(__half), hipMemcpyHostToDevice);

    printFP16Values<<<1, size>>>(d_fp16);
    hipDeviceSynchronize();

    delete[] h_fp16;
    hipFree(d_fp16);
    return 0;
}
